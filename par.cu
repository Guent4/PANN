#include "hip/hip_runtime.h"
// Compile:         gcc -Wall par.c -lm
// Run:             ./a.out <features> <N> <eta> <testSize> <num_layers> <layer1> <layer2> ...
// Note that regardless if what is put for the last layer, program will overwrite last layer to have size 1

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <float.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "matrix.h"

#define UINT_DIV_CEIL(X,Y) (1 + (((X) - 1) / (Y)))
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define BILLION 1000000000

#define TOTAL 8200
// 1d block size
#define BLOCK_SIZE 256

//ANN method
void testAccuracy(int testSize);
Matrix *feedForward(Matrix *in);
void backPropagation(Matrix *estimation);
void readInXY(int starting, int ending, Matrix *inputs, Matrix *outputs);
void initializeMatrices();
void freeMatrices();
uint64_t get_dt(struct timespec *start, struct timespec *end);

void printVector(float *vector, int len);

// cuda
__global__ void cuda_matirxElementSigmoid(float* A, int rows, int cols);

static hipblasHandle_t handle;
static hipblasStatus_t stat;

static int N;
static int FEATURES;
static int NUM_LAYERS;
static int *LAYER_SIZES;
static float ETA = 0.005;


static Matrix *XTS;
static Matrix *YTS;
static Matrix **WTS;
static Matrix **ZTS;

int main(int argc, char **argv)
{
    srand(time(NULL));
    FEATURES = (argc > 1) ? strtol(argv[1], NULL, 10) : 5;
    N = (argc > 2) ? strtol(argv[2], NULL, 10) : 5;
    ETA = (argc > 3) ? atof(argv[3]) : 0.01;
    int testSize = (argc > 4) ? strtol(argv[4], NULL, 10) : 100;
    NUM_LAYERS = (argc > 5) ? strtol(argv[5], NULL, 10) : 3;

    printf("eta %f\n", ETA);

    // fill in the layer sizes
    LAYER_SIZES = (int *)malloc(NUM_LAYERS * sizeof(int));

    for (int i = 0; i < NUM_LAYERS; i++) {
        LAYER_SIZES[i] = (argc > 6+i) ? strtol(argv[6+i], NULL, 10) : 10;
    }
    LAYER_SIZES[NUM_LAYERS - 1] = 1; // This has to be 1

    // init cublas
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        exit(1);
    }

    initializeMatrices();

    printf("test accuracy\n");
    testAccuracy(testSize);
    // printMatrix(WTS[1]);

    struct timespec start, end; //timestamps
    uint64_t total_ff = 0;
    uint64_t total_bp = 0;

    for (int outer = 0; outer < 1; outer++) {

        for (int iter = 0; iter < (TOTAL - testSize)/N; iter++) {
            // Retrieve data from csv
            readInXY(iter*N, iter*N + N, XTS, YTS);

            clock_gettime(CLOCK_MONOTONIC, &start);
            Matrix *out = feedForward(XTS);
            clock_gettime(CLOCK_MONOTONIC, &end);
            total_ff += get_dt(&start, &end);

            clock_gettime(CLOCK_MONOTONIC, &start);
            backPropagation(out);
            clock_gettime(CLOCK_MONOTONIC, &end);

            total_bp += get_dt(&start, &end);

            // printf("\n\n\n");
            if (iter % 20 == 0) {
                testAccuracy(testSize);
                // printMatrix(WTS[2]);
            }
            // printMatrix(WTS[1]);

            freeMatrix(out);
        }
    }

    float rt = (float)(total_bp + total_ff);
    printf("Feed Forward: %f%%, Back prop %f%%\n", 100*total_ff/rt, 100*total_bp/rt);


    freeMatrices();

    free(LAYER_SIZES);

}



// starting is included; ending is not
void readInXY(int starting, int ending, Matrix *inputs, Matrix *outputs)
{
    char buffer[2048];
    char *record, *line;
    int i, j;


    FILE* fstream = fopen("./dating/temp.csv", "r");

    if (fstream == NULL) {
        printf("\n file opening failed ");
        exit(1);
    }

    i = -1;     // Starts at -1 to account for row of column headers
    while((line = fgets(buffer, sizeof(buffer), fstream)) != NULL) {
        // Only include interested
        if (i >= starting && i < ending) {
            record = strtok(line, ",");

            // Put each token in the right location (X or Y)
            j = 0;
            while (record != NULL) {
                if (j == 0) {
                    outputs->m[IDXM(outputs, i-starting, 0)] = atof(record);
                } else {
                    inputs->m[IDXM(inputs, i-starting, j-1)] = atof(record);
                }

                j++;
                record = strtok(NULL, ",");
            }
        }

        i++;
    }
    fclose(fstream);

    // printMatrixMatlab(XTS);
}


void testAccuracy(int testSize)
{
    // Get test data
    Matrix *testX = newMatrix(testSize, FEATURES);
    Matrix *testY = newMatrix(testSize, 1);

    // Retrieve test data from csv
    readInXY(TOTAL-testSize, TOTAL, testX, testY);

    // Get the output
    Matrix *testOut = feedForward(testX);

    // Get the error
    Matrix *delta = matrixMatrixElementSub(testOut, testY);

    Matrix *trans = matrixTranspose(delta);
    printMatrix(trans);
    freeMatrix(trans);

    float error = matrixReduceSumPow(delta, 2);
    printf("Error: %f\n", error);

    freeMatrix(delta);
    freeMatrix(testOut);
    freeMatrix(testY);
    freeMatrix(testX);
}


Matrix *feedForward(Matrix *in)
{
    int wts_max = 0; //find max number of elements
    int max_cols = in->cols;
    for (int layer = 0; layer < NUM_LAYERS; layer++) {
        int tmp = WTS[layer]->cols*WTS[layer]->rows;
        wts_max = ( tmp > wts_max) ? tmp : wts_max;

        max_cols = (WTS[layer]->cols > max_cols) ? WTS[layer]->cols : max_cols;
    }

    float *dev_wts;
    float *dev_in;
    float *dev_z;
    float *dev_z_trans;
    hipMalloc((void**)&dev_wts, wts_max*sizeof(float));
    hipMalloc((void**)&dev_in, in->rows*max_cols*sizeof(float));
    hipMalloc((void**)&dev_z, in->rows*max_cols*sizeof(float));
    hipMalloc((void**)&dev_z_trans, in->rows*max_cols*sizeof(float));

    const float alpha = 1;
    const float beta = 0;


    const int in_rows = in->rows;
    int in_cols = in->cols;

    int wts_cols = WTS[0]->cols;
    int wts_rows = WTS[0]->rows;

    // this will load in transposed
    hipblasSetMatrix(in->cols, in->rows, sizeof(float),
            in->m, in->cols, dev_in, in->cols);

    for (int layer = 0; layer < NUM_LAYERS; layer++) {

        wts_cols = WTS[layer]->cols;
        wts_rows = WTS[layer]->rows;

        if (in_cols != wts_rows) {
            printf("Error! Dimension mismatch in feedforward\n");
            exit(1);
        }

        // Load WTS[layer]  transposed
        hipblasSetMatrix(wts_cols, wts_rows, sizeof(float),
                WTS[layer]->m, wts_cols, dev_wts, wts_cols);


        hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
            in_rows, wts_cols, in_cols, &alpha, dev_in, in_cols,
            dev_wts, wts_cols, &beta, dev_z, in_rows);

        // only apply sigmoid if not last layer
        if (layer == NUM_LAYERS - 1) // last output layer
            break;

        // Multiply Z with W to get S
        //z = matrixMatrixMultiply(in, WTS[layer]);
        //z = newMatrix(in_rows, wts_cols);

        // now dev_z is in_rows x wts_cols (stored in row ordering on device)


        //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        //dim3 dimGrid(UINT_DIV_CEIL(wts_cols, dimBlock.x), UINT_DIV_CEIL(in_rows , dimBlock.y));

        int blocks = UINT_DIV_CEIL((wts_cols*in_rows), BLOCK_SIZE);

        //printf("Launching kernel with dim y: %d, dim x: %d\n", UINT_DIV_CEIL(wts_cols, dimBlock.x), UINT_DIV_CEIL(in_rows , dimBlock.y));


        cuda_matirxElementSigmoid<<<blocks, BLOCK_SIZE>>>(dev_z, in_rows, wts_cols);



        // stupid transpose to put it into col ordering
        // dev_z is dev_z[in_rows][wts_cols] (stored in row ordering on device)
        // now dev_z_trans is wts_cols x in_rows (stored in row ordering on device)


        hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, wts_cols, in_rows, &alpha,
                dev_z, in_rows, &beta, dev_z, wts_cols, dev_z_trans, wts_cols);

        // now dev_z is WTS[layer]->cols x in->rows (stored in row ordering on device)


        hipMemcpy(ZTS[layer]->m, dev_z_trans, in_rows*wts_cols*sizeof(float), hipMemcpyDeviceToHost); //eventually make async



        // Apply activation function to S to get Z
        //matrixElementApply(z, sigmoid);

        // Save Z because this is sigmoid(S) and is needed in back propagation
        //ZTS[layer] = z;

        // Update values for next iteration
        //in = z;

        //swap
        float *tmp = dev_in;
        dev_in = dev_z_trans;
        dev_z_trans = tmp;

        // update col dims
        in_cols = wts_cols;
    }

    Matrix *z = newMatrix(in_rows, WTS[NUM_LAYERS-1]->cols);

    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, wts_cols, in_rows, &alpha, dev_z,
        in_rows, &beta, dev_z, wts_cols, dev_z_trans, wts_cols);

    // now dev_z is WTS[layer]->cols x in->rows (stored in row ordering on device)
    hipMemcpy(z->m, dev_z_trans, in_rows*wts_cols*sizeof(float), hipMemcpyDeviceToHost); //eventually make async


    hipFree (dev_wts);
    hipFree (dev_in);
    hipFree (dev_z);
    hipFree (dev_z_trans);

    // feed through last layer
    return z;
}



void backPropagation(Matrix *estimation)
{

    // Backprop
    Matrix **D = (Matrix **)malloc(NUM_LAYERS * sizeof(Matrix *));

    for (int layer = NUM_LAYERS - 1; layer >= 0; layer--) {


        if (layer == NUM_LAYERS - 1) {
            Matrix *Dtrans = matrixMatrixElementSub(estimation, YTS);
            D[layer] = matrixTranspose(Dtrans);
            freeMatrix(Dtrans);
        } else {

            matrixElementApply(ZTS[layer], sigmoidDerivWhenAlreadyHaveSigmoid);
            Matrix *F = matrixTranspose(ZTS[layer]);

            Matrix *WD = matrixMatrixMultiply(WTS[layer + 1], D[layer + 1]);

            D[layer] = matrixMatrixElementMultiply(F, WD);

            freeMatrix(WD);
            freeMatrix(F);
        }
    }

    // Weight Updates
    for (int layer = 0; layer < NUM_LAYERS; layer++) {
        Matrix *DZ;
        if (layer == 0) {
            DZ = matrixMatrixMultiply(D[layer], XTS);
        } else {
            DZ = matrixMatrixMultiply(D[layer], ZTS[layer - 1]);
        }

        Matrix *wUpdates = matrixTranspose(DZ);
        float neta = -1*ETA;
        matrixElementApplyArg(wUpdates, multByConst, &neta);
        WTS[layer] = matrixMatrixElementAdd(WTS[layer], wUpdates);

        freeMatrix(wUpdates);
        freeMatrix(DZ);
    }

    // Free temporary matrices
    for (int i = 0; i < NUM_LAYERS; i++) {
        freeMatrix(D[i]);
    }
}



void initializeMatrices()
{

	// Create input
    XTS = newMatrix(N, FEATURES);

	// Create output
    YTS = newMatrix(N, 1);

    // Create weight matrices
    WTS = (Matrix **)malloc(NUM_LAYERS * sizeof(Matrix **));
    for (int i = 0; i < NUM_LAYERS; i++) {
        int numRows = (i == 0) ? FEATURES : LAYER_SIZES[i-1];

        WTS[i] = newMatrix(numRows, LAYER_SIZES[i]);

        // The in->firstHidden and lastHidden->out have weights of 1
        if (i == 0) {
            matrixElementApply(WTS[i], setTo0);
        } else if (i == NUM_LAYERS-1) {
            matrixElementApply(WTS[i], setTo1);
            //WTS[i]->m[IDXM(WTS[i],0,0)] = 1;
        } else {
            matrixElementApply(WTS[i], setToRand);
        }

    }

    // Create S matrices
    ZTS = (Matrix **)malloc((NUM_LAYERS - 1) * sizeof(Matrix **));
    for (int i = 0; i < NUM_LAYERS - 1; i++) {
        ZTS[i] = newMatrix(N, LAYER_SIZES[i]);
    }
}


void freeMatrices()
{
    // Free X, Y
    freeMatrix(XTS);
    freeMatrix(YTS);

    // Free weights matrix
    for (int i = 0; i < NUM_LAYERS; i++) {
        freeMatrix(WTS[i]);
    }
    free(WTS);

    // Free Z matrix
    for (int i = 0; i < NUM_LAYERS - 1; i++) {
        freeMatrix(ZTS[i]);
    }
    free(ZTS);
}


void printVector(float *vector, int len)
{
	printf("------------------------------------\n");
	int i;
	for (i = 0; i < len; i++) {
		printf("%f\n", vector[i]);
	}
	printf("------------------------------------\n");
}



uint64_t get_dt(struct timespec *start, struct timespec *end)
{
    return BILLION*(end->tv_sec - start->tv_sec) + (end->tv_nsec - start->tv_nsec);
}


__global__ void cuda_matirxElementSigmoid(float* A, int rows, int cols)
{
    int tid = blockIdx.x *blockDim.x + threadIdx.x;


    if (tid < rows*cols)
    {
        A[IDX2C(tid%rows, tid/rows, rows)] = 1.0/(1.0 + expf(-1*IDX2C(tid%rows, tid/rows, rows)));
    }

}
